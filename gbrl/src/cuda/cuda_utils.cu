#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "cuda_utils.h"
#include "cuda_types.h"



void get_grid_dimensions(int n_elements, int& blocks, int& threads_per_block) {
    const int max_threads_per_block = 256; // You can adjust this based on your GPU

    if (n_elements < WARP_SIZE) {
        threads_per_block = WARP_SIZE;
        blocks = 1;
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    blocks = n_elements / max_threads_per_block + 1;

    threads_per_block = 1 << static_cast<int>(ceilf(log2f(static_cast<float>(n_elements) / blocks)));
    threads_per_block = (threads_per_block > deviceProp.maxThreadsPerBlock) ? deviceProp.maxThreadsPerBlock : threads_per_block;
}